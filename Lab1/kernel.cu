#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <string>
#include <iostream>
#include <sstream>
#include <fstream>

#define MAX_THREADS_X_IN_BLOCK_SIZE 1024
#define BLOCK_X_AMOUNT 1024
#define MAX_CHUNK_ARRAY_SIZE (MAX_THREADS_X_IN_BLOCK_SIZE * BLOCK_X_AMOUNT)

using namespace std;

hipError_t arrayCalculations(double* c, const double* a, const double* b, unsigned int size);
unsigned int loadFileToArray(std::ifstream &file, double* a, unsigned int sizeToRead);
int saveArrayToFile(double* c, string fileName, unsigned int sizeToWrite);

__global__ void calculateKernel(double* c, const double* a, const double* b, long long size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < size; i += stride)
        c[i] = (a[i] + b[i]) * 4 + 5;
}

int main()
{
    // 2 ** 20 - max possible array chunk
    const unsigned int arraySize = MAX_CHUNK_ARRAY_SIZE;

    double a[arraySize] = {};
    double b[arraySize] = {};
    double c[arraySize] = {};

    //long long calculationLength = std::pow(2, 33);
    long long calculationLength = (long long)std::pow(2, 21);
    string fileName1 = "..\\test_file1_floats.txt";
    string fileName2 = "..\\test_file2_floats.txt";
    string fileNameResult = "..\\result.txt";

    long long currentFilePossition = 0;
    int numberOfIterations = (int) (calculationLength / MAX_CHUNK_ARRAY_SIZE);

    hipError_t cudaStatus;
    std::ifstream file1(fileName1);
    std::ifstream file2(fileName2);

    for (size_t i = 0; i < numberOfIterations; i++)
    {
        // setArrays from files
        unsigned int setSize1 = loadFileToArray(file1, a, arraySize);
        unsigned int setSize2 = loadFileToArray(file2, b, arraySize);

        if (setSize1 == 0 || setSize2 == 0)
        {
            printf("Unable to open file for reading.");
            return 1;
        }
        if (setSize1 != setSize2)
        {
            printf("Lines in files should be of the same length.");
            return 1;
        }

        // Add vectors in parallel.
        cudaStatus = arrayCalculations(c, a, b, setSize1);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "arrayCalculations failed!");
            return 1;
        }

        // save result array to file
        int result = saveArrayToFile(c, fileNameResult, setSize1);

        if (result == -1)
        {
            printf("Unable to open file for writing.");
            return 1;
        }

        currentFilePossition += setSize1;
    }

    file1.close();
    file2.close();

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// recieves files with values from [0, 100)
unsigned int loadFileToArray(std::ifstream &file, double* a, unsigned int sizeToRead) {

    std::string line;

    if (!file.is_open())
    {
        return 0;
    }

    unsigned int loop = 0;
    double currentValue = 0;

    while (!file.eof() && loop < sizeToRead)
    {
        std::getline(file, line);
        currentValue = std::stod(line) / 100;
        a[loop] = currentValue;
        loop++;
    }

    return loop;
}

int saveArrayToFile(double* c, string fileName, unsigned int sizeToWrite)
{
    ofstream file(fileName, std::ios_base::app);
    
    if (!file.is_open())
    {
        return -1;
    }

    for (unsigned int count = 0; count < sizeToWrite; count++) {
        std::ostringstream ss;
        ss << c[count];
        ss << "\n";
        file << ss.str();
    }
    file.close();
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t arrayCalculations(double* c, const double* a, const double* b, unsigned int size)
{
    double* dev_a = 0;
    double* dev_b = 0;
    double* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output).
    cudaStatus = hipMalloc((void**)& dev_c, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)& dev_a, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)& dev_b, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    int numBlocks = (size + MAX_THREADS_X_IN_BLOCK_SIZE - 1) / MAX_THREADS_X_IN_BLOCK_SIZE;;
    int numThreads = MAX_THREADS_X_IN_BLOCK_SIZE;
    if (size < MAX_THREADS_X_IN_BLOCK_SIZE)
    {
        numBlocks = 1;
        numThreads = size;
    }
    calculateKernel <<<numBlocks, numThreads>>> (dev_c, dev_a, dev_b, size);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "arrayCalculations launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching arrayCalculations!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
